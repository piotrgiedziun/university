/**
* parms
* top left bottom right
*
* example:
* 0 0 10 10
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <math.h>

#define MATRIX_WIDTH				2016
#define MATRIX_HEIGHT				2016
#define MATRIX_SIZE					MATRIX_WIDTH*MATRIX_HEIGHT
#define TILE_WIDTH					48//48
#define TILE_HEIGHT					48//48
#define ELEMENTS_PER_THREAD_Y		16
#define ROWS_PER_THREAD				2
#define ELEMENTS_PER_MATRIX_ROW		MATRIX_WIDTH*TILE_HEIGHT
#define TILE_SIZE					TILE_WIDTH*TILE_HEIGHT
#define INIT_THREADS_PER_BLOCK		256
#define INIT_ELEMENTS_PER_THREAD	90 
#define INIT_ELEMENTS_PER_BLOCK		INIT_ELEMENTS_PER_THREAD*INIT_THREADS_PER_BLOCK

#define TILE_DIM 32
#define BLOCK_ROWS 8

// window struct
struct WINDOW {
	int top;
	int left;
	int bottom;
	int right;
} show_window = { 0, 0, 10, 10 };

/*
* Matrix transpose - device function 
* ----------------------------------------
* Description:
* 	this function transpose matrix element block by block
*	and line by line
* Parms:
* 	- idata (int*)	- input matrix
*	- odata (int*)	- output matix (allocated alrea)
*/
__global__ void transpose_matrix(int* idata, int* odata) {
	// allocate shared memory
	__shared__ int tile[TILE_HEIGHT][TILE_WIDTH];
	
	// estimate current position
	int inputPos = blockIdx.y*ELEMENTS_PER_MATRIX_ROW + blockIdx.x*TILE_WIDTH;
	int outputPos = blockIdx.x*ELEMENTS_PER_MATRIX_ROW + blockIdx.y*TILE_WIDTH;
	int coPos;
	int startPos = threadIdx.y*ROWS_PER_THREAD;
	
	// copy to shared memory
	for(int j=0; j <= ROWS_PER_THREAD-1; j++) {
		coPos = startPos*MATRIX_WIDTH + (MATRIX_WIDTH*j);
		for(int i=threadIdx.x; i < TILE_WIDTH; i+=ELEMENTS_PER_THREAD_Y) {
			tile[startPos+j][i] = idata[inputPos+i+coPos];	
		}
	}

	__syncthreads();

	// copy back to global memory
	for(int j=0; j <= ROWS_PER_THREAD-1; j++) {
		coPos = startPos*MATRIX_WIDTH + (MATRIX_WIDTH*j);
		for(int i=threadIdx.x; i < TILE_WIDTH; i+=ELEMENTS_PER_THREAD_Y) {
			odata[outputPos+i+coPos] = tile[i][startPos+j];
		}
	}

}

/*
* Initialize matrix - device function 
* ----------------------------------------
* Description:
* 	this function initialze matrix INIT_ELEMENTS_PER_THREAD elements by elements
* Parms:
* 	- idata (int*)	- matrix
*	- size (int)	- matrix size (number of elements)
*/
__global__ void init_matrix( int *idata, int size ) {
	int	elements_count;
	int start_id = (threadIdx.x * INIT_ELEMENTS_PER_THREAD );
	
	// set elements count
	if ( start_id+INIT_ELEMENTS_PER_THREAD > size ) {
		elements_count = size-start_id;
	}else{
		elements_count = INIT_ELEMENTS_PER_THREAD;
	}
	
	// set elementsPerThread elements
	int value = start_id + ( blockIdx.x * INIT_ELEMENTS_PER_BLOCK );
	for( int i=0; i < elements_count; i++ )
		idata[value+i] = value+i;
}

__global__ void init_matrix_zero( int *idata, int size ) {
	int	elements_count;
	int start_id = (threadIdx.x * INIT_ELEMENTS_PER_THREAD );
	
	// set elements count
	if ( start_id+INIT_ELEMENTS_PER_THREAD > size ) {
		elements_count = size-start_id;
	}else{
		elements_count = INIT_ELEMENTS_PER_THREAD;
	}
	
	// set elementsPerThread elements
	int value = start_id + ( blockIdx.x * INIT_ELEMENTS_PER_BLOCK );
	for( int i=0; i < elements_count; i++ )
		idata[value+i] = 0;
}

/*
* Print matrix window - host function
* ----------------------------------------
* Description:
* 	this function copy data form device
* 	and print output to screen
* 	host matrix size has to be the same as device matrix size
* Pamrs:
*	- h_matrix (int*) - host matrix
*	- d_matrix (int*) - device matrix
*	- start_height, start_width, end_height, end_width (int) - displaying area
*/
void print_matrix_window(int* h_matrix, int* d_matrix, int start_height, int start_width, int end_height, int end_width ) {
	hipMemcpy( h_matrix, d_matrix, sizeof(int)*MATRIX_SIZE, hipMemcpyDeviceToHost);
	printf("--------------------\n");
	for(int y=start_height; y < end_height; y++) {
		for(int x=start_width; x < end_width; x++) {
			printf("%d\t", h_matrix[x + ( y * MATRIX_WIDTH ) ]);
		}
		printf("\n");
	}
}

/*
* return GB per second for given data
*/
double GBperSec(float runtime, double bytes) {
	return 100*(bytes/1073741824)/runtime;
}

int main( int argc, char *argv[]) {
	if( argc > 4) {
		show_window.top=atoi(argv[1]);
		show_window.left=atoi(argv[2]);
		show_window.bottom=atoi(argv[3]);
		show_window.right=atoi(argv[4]);
	}
	
	int *d_idata, *d_odata, *h_matrix, blocks_count;
	hipEvent_t init_start, init_end;
	float init_time;
	// create init events
	hipEventCreate(&init_start);
	hipEventCreate(&init_end);
	hipEvent_t transpose_start, transpose_end;
	float transpose_time;
	// create transpose events
	hipEventCreate(&transpose_start);
	hipEventCreate(&transpose_end);

	// size validation
	if ( MATRIX_WIDTH % TILE_WIDTH != 0
	 	|| MATRIX_HEIGHT % TILE_HEIGHT != 0) {
		printf("Invalid matrix size\n");
		return 1;
	}

	hipMalloc( (void**)&d_idata, sizeof(int) * MATRIX_SIZE );
	hipMalloc( (void**)&d_odata, sizeof(int) * MATRIX_SIZE );
	h_matrix = new int[ MATRIX_SIZE ];
	
	// matrix initialization
	// set data to global memory
	blocks_count = ceil(((MATRIX_SIZE+INIT_THREADS_PER_BLOCK-1)/INIT_THREADS_PER_BLOCK)/INIT_ELEMENTS_PER_THREAD);
	
	hipEventRecord(init_start, 0);
 
	init_matrix<<< blocks_count, INIT_THREADS_PER_BLOCK >>>(d_idata, MATRIX_SIZE);
	init_matrix_zero<<< blocks_count, INIT_THREADS_PER_BLOCK >>>(d_odata, MATRIX_SIZE);
	
	hipEventRecord(init_end, 0);
	hipEventSynchronize(init_end);
	hipEventElapsedTime(&init_time, init_start, init_end);
	
	printf("init matrix:\n\tblocks count: %d\n\tthreads per block: %d\n\telements per thread: %d\n\ttime: %f ms\n\tspeed: %lf GB/s\n",
		blocks_count, INIT_THREADS_PER_BLOCK, INIT_ELEMENTS_PER_THREAD, init_time, GBperSec(init_time, MATRIX_SIZE*sizeof(int)));
	print_matrix_window(h_matrix, d_idata, show_window.top, show_window.left, show_window.bottom, show_window.right);
	
	dim3 blocks(MATRIX_HEIGHT/TILE_HEIGHT, MATRIX_WIDTH/TILE_HEIGHT);
	dim3 threads(ELEMENTS_PER_THREAD_Y, TILE_WIDTH/ROWS_PER_THREAD);

	hipEventRecord(transpose_start, 0);

	transpose_matrix<<< blocks, threads >>>(d_idata, d_odata);
	
	hipEventRecord(transpose_end, 0);
	hipEventSynchronize(transpose_end);
	hipEventElapsedTime(&transpose_time, transpose_start, transpose_end);

	printf("\ntranspose matrix:\n\tblocks count x: %d\n\tblocks count y: %d\n\tthreads count x: %d\n\tthreads count y: %d\n\ttime: %f ms\n\tspeed: %lf GB/s\n",
		MATRIX_HEIGHT/TILE_HEIGHT, MATRIX_WIDTH/TILE_HEIGHT, ELEMENTS_PER_THREAD_Y, TILE_WIDTH/ROWS_PER_THREAD,
	 transpose_time, GBperSec(transpose_time, MATRIX_SIZE*sizeof(int)));
	print_matrix_window(h_matrix, d_odata, show_window.top, show_window.left, show_window.bottom, show_window.right);
		
	// clear memory allocation
	hipFree( d_idata );
	hipFree( d_odata );
	delete [] h_matrix;
	return 0;
}
