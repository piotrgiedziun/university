
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <math.h>

#define MATRIX_WIDTH				2025
#define MATRIX_HEIGHT				2025
#define MATRIX_SIZE					MATRIX_WIDTH*MATRIX_HEIGHT
#define TILE_WIDTH					45
#define TILE_HEIGHT					45
#define INIT_THREADS_PER_BLOCK		256
#define INIT_ELEMENTS_PER_THREAD	90 
#define INIT_ELEMENTS_PER_BLOCK		INIT_ELEMENTS_PER_THREAD*INIT_THREADS_PER_BLOCK

/*
* Matrix transpose - device function 
* ----------------------------------------
* Description:
* 	this function transpose matrix element block by block
*	and line by line
* Parms:
* 	- idata (int*)	- input matrix
*	- odata (int*)	- output matix (allocated alrea)
*/
__global__ void transpose_matrix(int* idata, int* odata, int block_per_row) {
	// allocate shared memory
	__shared__ int shared_input[TILE_HEIGHT][TILE_WIDTH];
	__shared__ int shared_output[TILE_WIDTH][TILE_HEIGHT];
	
	// estimate current position
	int line = threadIdx.x;
	int block = blockIdx.x;
	
	// local data from global memory (idata) to shared memory (shared_input)
	int block_row = (block/block_per_row);
	int block_column = block % block_per_row;
	
	// cpy
	int block_start_pos =(block_row * (TILE_HEIGHT*MATRIX_WIDTH) + (block_column*TILE_WIDTH + (line*MATRIX_WIDTH))); //(block_row*TILE_HEIGHT+(line*MATRIX_WIDTH));
	//int block_shift = TILE_WIDTH*block;
	
	for(int i=0; i < TILE_WIDTH; i++) {
		shared_input[line][i] = idata[block_start_pos+i];
	}
	
	// single transposition blok
	for(int i=0;i<TILE_WIDTH;i++)
	{
		shared_output[i][line] = shared_input[line][i];
	}

__syncthreads();
	if(block == 0) {
		block_start_pos = (block_column * (TILE_WIDTH*MATRIX_HEIGHT) + (block_row*TILE_HEIGHT + (line*MATRIX_HEIGHT)));
		for(int i=0; i<TILE_HEIGHT; i++) {
			odata[block_start_pos+i] = shared_output[line][i];
 		}
	//v 2 memcpy
	//memcpy( odata, shared_output, TILE_HEIGHT * sizeof(int) );  // TODO: napisac poprawnie zapisywanie wynikow w macierzy wynikowej
	}

}

/*
* Initialize matrix - device function 
* ----------------------------------------
* Description:
* 	this function initialze matrix INIT_ELEMENTS_PER_THREAD elements by elements
* Parms:
* 	- idata (int*)	- matrix
*	- size (int)	- matrix size (number of elements)
*/
__global__ void init_matrix( int *idata, int size ) {
	int	elements_count;
	int start_id = (threadIdx.x * INIT_ELEMENTS_PER_THREAD );
	
	// set elements count
	if ( start_id+INIT_ELEMENTS_PER_THREAD > size ) {
		elements_count = size-start_id;
	}else{
		elements_count = INIT_ELEMENTS_PER_THREAD;
	}
	
	// set elementsPerThread elements
	int value = start_id + ( blockIdx.x * INIT_ELEMENTS_PER_BLOCK );
	for( int i=0; i < elements_count; i++ )
		idata[value+i] = value+i;
}

/*
* Print matrix window - host function
* ----------------------------------------
* Description:
* 	this function copy data form device
* 	and print output to screen
* 	host matrix size has to be the same as device matrix size
* Pamrs:
*	- h_matrix (int*) - host matrix
*	- d_matrix (int*) - device matrix
*	- start_height, start_width, end_height, end_width (int) - displaying area
*/
void print_matrix_window(int* h_matrix, int* d_matrix, int start_height, int start_width, int end_height, int end_width ) {
	hipMemcpy( h_matrix, d_matrix, sizeof(int)*MATRIX_SIZE, hipMemcpyDeviceToHost);
	printf("--------------------\n");
	for(int y=start_height; y < end_height; y++) {
		for(int x=start_width; x < end_width; x++) {
			printf("%d\t", h_matrix[x + ( y * MATRIX_WIDTH ) ]);
		}
		printf("\n");
	}
}

/*
* return GB per second for given data
*/
double GBperSec(float runtime, double bytes) {
	return 100*(bytes/1073741824)/runtime;
}

int main(void) {
	int number_of_device,device_number;
	hipGetDeviceCount(&number_of_device);
	if(number_of_device >0){
		for(device_number=0;device_number<number_of_device; device_number++){
			hipDeviceProp_t device_prop;
			hipGetDeviceProperties(&device_prop, device_number);
			printf("number of device: %d prop: %s\n",device_number, device_prop.name);
		}
	}

	int *d_idata, *d_odata, *h_matrix, blocks_count, block_per_row;
	hipEvent_t init_start, init_end;
	float init_time;
	// create init events
	hipEventCreate(&init_start);
	hipEventCreate(&init_end);
	hipEvent_t transpose_start, transpose_end;
	float transpose_time;
	// create transpose events
	hipEventCreate(&transpose_start);
	hipEventCreate(&transpose_end);

	// size validation
	if ( MATRIX_WIDTH % TILE_WIDTH != 0
	 	|| MATRIX_HEIGHT % TILE_HEIGHT != 0) {
		printf("Invalid matrix size\n");
		return 1;
	}

	hipMalloc( (void**)&d_idata, sizeof(int) * MATRIX_SIZE );
	hipMalloc( (void**)&d_odata, sizeof(int) * MATRIX_SIZE );
	h_matrix = new int[ MATRIX_SIZE ];
	
	// matrix initialization
	// set data to global memory
	blocks_count = ceil(((MATRIX_SIZE+INIT_THREADS_PER_BLOCK-1)/INIT_THREADS_PER_BLOCK)/INIT_ELEMENTS_PER_THREAD);
	
	hipEventRecord(init_start, 0);
 
	init_matrix<<< blocks_count, INIT_THREADS_PER_BLOCK >>>(d_idata, MATRIX_SIZE);
	
	hipEventRecord(init_end, 0);
	hipEventSynchronize(init_end);
	hipEventElapsedTime(&init_time, init_start, init_end);
	
	printf("init matrix:\n\tblocks count: %d\n\tthreads per block: %d\n\telements per thread: %d\n\ttime: %f ms\n\tspeed: %lf GB/s\n",
		blocks_count, INIT_THREADS_PER_BLOCK, INIT_ELEMENTS_PER_THREAD, init_time, GBperSec(init_time, MATRIX_SIZE*sizeof(int)));
	print_matrix_window(h_matrix, d_idata, 0, 0, 10, 10);
	
	blocks_count = MATRIX_SIZE / (TILE_WIDTH*TILE_HEIGHT);
	block_per_row = MATRIX_WIDTH/TILE_WIDTH;
	
	hipEventRecord(transpose_start, 0);

	transpose_matrix<<< blocks_count, TILE_HEIGHT >>>(d_idata, d_odata, block_per_row);
	
	hipEventRecord(transpose_end, 0);
	hipEventSynchronize(transpose_end);
	hipEventElapsedTime(&transpose_time, transpose_start, transpose_end);

	printf("\ntranspose matrix:\n\tblocks count: %d\n\tthreads per block: %d\n\tblock per row: %d\n\ttime: %f ms\n\tspeed: %lf GB/s\n",
		blocks_count, TILE_HEIGHT, block_per_row, transpose_time, GBperSec(transpose_time, MATRIX_SIZE*sizeof(int)));
	print_matrix_window(h_matrix, d_odata, 0, 0, 10, 10);
		
	// clear memory allocation
	hipFree( d_idata );
	hipFree( d_odata );
	delete [] h_matrix;
	return 0;
}
