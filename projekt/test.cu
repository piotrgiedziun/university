
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <math.h>

#define MATRIX_WIDTH				2025
#define MATRIX_HEIGHT				2025
#define MATRIX_SIZE					4100625
#define TAIL_WIDTH					45
#define TAIL_HEIHGT					45
#define INIT_THREADS_PER_BLOCK		256
#define INIT_ELEMENTS_PER_THREAD	90 

// simple set to testing
// #define MATRIX_WIDTH				10
// #define MATRIX_HEIGHT			10
// #define MATRIX_SIZE				100
// #define TAIL_WIDTH				45
// #define TAIL_HEIHGT				45
// #define INIT_THREADS_PER_BLOCK	2
// #define INIT_ELEMENTS_PER_THREAD	2

/*
* Matrix transpose - device function 
* ----------------------------------------
* Description:
* 	this function transpose matrix element block by block
*	and line by line
* Parms:
* 	- idata (int*)	- input matrix
*	- odata (int*)	- output matix (allocated alrea)
*/
__global__ void transpose_matrix(int* idata, int* odata) {
	// allocate shared memory
	__shared__ int shared_input[TAIL_HEIHGT][TAIL_WIDTH];
	__shared__ int shared_output[TAIL_HEIHGT][TAIL_WIDTH];
	
	// estimate current position
	int line = threadIdx.x;
	int blok = blockIdx.x;
	
	// local data from global memory (idata) to shared memory (shared_input)
	
	// transose line to shared memory (shared_output)
	
	// export converted data to output
	//__syncthreads();
}

/*
* Initialize matrix - device function 
* ----------------------------------------
* Description:
* 	this function initialze matrix INIT_ELEMENTS_PER_THREAD elements by elements
* Parms:
* 	- idata (int*)	- matrix
*	- size (int)	- matrix size (number of elements)
*/
__global__ void init_matrix( int *idata, int size ) {
	int	elements_count,
		start_id = threadIdx.x + (blockIdx.x * INIT_THREADS_PER_BLOCK );
	
	// set elements count
	if ( start_id+INIT_ELEMENTS_PER_THREAD > size ) {
		elements_count = size-start_id;
	}else{
		elements_count = INIT_ELEMENTS_PER_THREAD;
	}
	
	// set elementsPerThread elements
	for( int i=start_id; i < elements_count; i++ )
		idata[i] = i;
}

// slower version
__global__ void init_matrix_cpy( int *idata, int size ) {
	int init_values[INIT_ELEMENTS_PER_THREAD], // temp array of inital values
	 	elements_count,
		start_id = ( threadIdx.x + (blockIdx.x * blockDim.x) );
	
	// set elements count
	if ( start_id+INIT_ELEMENTS_PER_THREAD > size ) {
		elements_count = size-start_id;
	}else{
		elements_count = INIT_ELEMENTS_PER_THREAD;
	}
	
	// set elementsPerThread elements
	for( int i=0; i < elements_count; i++ )
		init_values[i] = i;
	
	// copy initial data to global memoory (idata)	
	 memcpy(idata+start_id, init_values, elements_count * sizeof(int));
}

/*
* Print matrix window - host function
* ----------------------------------------
* Description:
* 	this function copy data form device
* 	and print output to screen
* 	host matrix size has to be the same as device matrix size
* Pamrs:
*	- h_matrix (int*) - host matrix
*	- d_matrix (int*) - device matrix
*	- start_height, start_width, end_height, end_width (int) - displaying area
*/
void print_matrix_window(int* h_matrix, int* d_matrix, int start_height, int start_width, int end_height, int end_width ) {
	hipMemcpy( h_matrix, d_matrix, sizeof(int)*MATRIX_SIZE, hipMemcpyDeviceToHost);
	printf("--------------------\n");
	for(int y=start_height; y < end_height; y++) {
		for(int x=start_width; x < end_width; x++) {
			printf("%d\t", h_matrix[x + ( y * MATRIX_WIDTH ) ]);
		}
		printf("\n");
	}
}


double GBperSec(float runtime, double bytes) {
	return 100*(bytes/1073741824)/runtime;
}

int main(void) {
	int *d_idata, *d_odata, *h_matrix, blocks_count;
	hipEvent_t init_start, init_end;
	float init_time;
	// create init events
	hipEventCreate(&init_start);
	hipEventCreate(&init_end);
	hipEvent_t transpose_start, transpose_end;
	float transpose_time;
	// create transpose events
	hipEventCreate(&transpose_start);
	hipEventCreate(&transpose_end);

	// size validation
	if ( MATRIX_WIDTH % TAIL_WIDTH != 0
	 	|| MATRIX_HEIGHT % TAIL_HEIHGT != 0) {
		printf("Invalid matrix size\n");
		return 1;
	}

	hipMalloc( (void**)&d_idata, sizeof(int) * MATRIX_SIZE );
	hipMalloc( (void**)&d_odata, sizeof(int) * MATRIX_SIZE );
	h_matrix = new int[ MATRIX_SIZE ];
	
	// matrix initialization
	// set data to global memory
	blocks_count = ceil(((MATRIX_SIZE+INIT_THREADS_PER_BLOCK-1)/INIT_THREADS_PER_BLOCK)/INIT_ELEMENTS_PER_THREAD);
		
	hipEventRecord(init_start, 0); 
	init_matrix<<< blocks_count, INIT_THREADS_PER_BLOCK >>>(d_idata, MATRIX_SIZE);
	hipEventRecord(init_end, 0);
	hipEventSynchronize(init_end);
	hipEventElapsedTime(&init_time, init_start, init_end);
	
	printf("init matrix:\n\tblocks count: %d\n\tthreads per block: %d\n\telements per thread: %d\n\ttime: %f ms\n\tspeed: %lf GB/s\n",
		blocks_count, INIT_THREADS_PER_BLOCK, INIT_ELEMENTS_PER_THREAD, init_time, GBperSec(init_time, MATRIX_SIZE*sizeof(int)));
	print_matrix_window(h_matrix, d_idata, 0, 0, 10, 10);
	
	blocks_count = MATRIX_SIZE / (TAIL_WIDTH*TAIL_HEIHGT);
	
	hipEventRecord(transpose_start, 0);
	transpose_matrix<<< blocks_count, TAIL_HEIHGT >>>(d_idata, d_odata);
	hipEventRecord(transpose_end, 0);
	hipEventSynchronize(transpose_end);
	hipEventElapsedTime(&transpose_time, transpose_start, transpose_end);

	printf("\ntranspose matrix:\n\tblocks count: %d\n\tthreads per block: %d\n\ttime: %f ms\n\tspeed: %lf GB/s\n",
		blocks_count, TAIL_HEIHGT, transpose_time, GBperSec(transpose_time, MATRIX_SIZE*sizeof(int)));
	//print_matrix_window(h_matrix, d_odata, 0, 0, 10, 10);
		
	// clear memory allocation
	hipFree( d_idata );
	hipFree( d_odata );
	delete [] h_matrix;
	return 0;
}